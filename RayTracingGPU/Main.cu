#include "hip/hip_runtime.h"
#include <iostream>
#include <time.h>
#include "vec3.h"
#include "ray.h"
#include "hittable_list.h"
#include "sphere.h"
#include "camera.h"
#include <hiprand/hiprand_kernel.h>
#include "material.h"

#define checkCudaErrors(val) check_cuda((val), #val, __FILE__, __LINE__)

//hipError_t result,
void check_cuda(int result, char const* const func, const char* const file, int const line)
{
	if (result)
	{
		std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " << file << ":" << line << " '" << func << "' " << std::endl;
		// Make sure to call CUDA device reset before exist. For profiling tool to work.  
		hipDeviceReset();
		exit(99);
	}
}

// Render
__device__ color3 ray_color(const ray& r, hittable** world, hiprandState *local_rand_state)
{
	ray cur_ray = r; 
	point3 cur_attenuation = point3(1.0, 1.0, 1.0); 
	for (int i = 0; i < 50; i++)
	{
		hit_record rec;
		if ((*world)->hit(cur_ray, 1e-10, DBL_MAX, rec))
		{
			ray scattered; 
			point3 attenuation; 
			if (rec.mat_ptr->scatter(cur_ray, rec, attenuation, scattered, local_rand_state))
			{
				cur_attenuation *= attenuation;
				cur_ray = scattered;
			}
			else
			{
				return color3(0.0, 0.0, 0.0); 
			}
		}
		else
		{
			vec3 unit_direction = unit_vector(cur_ray.direction());
			double t = 0.5 * (unit_direction.y() + 1.0);
			vec3 color = (1.0 - t) * vec3(1.0, 1.0, 1.0) + t * vec3(0.5, 0.7, 1.0); 
			return cur_attenuation * color;
		}
	}
	// exceeded recussion depth 
	return vec3(0.0, 0.0, 0.0); 
}

__global__ void rand_init(hiprandState *rand_state)
{
	if (threadIdx.x == 0 && threadIdx.y == 0)
		hiprand_init(1984, 0, 0, rand_state); 
}


__global__ void render_init(int max_x, int max_y, hiprandState* rand_state)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	if ((i >= max_x) || (j >= max_y)) return;
	int pixel_index = (j * max_x + i);
	// Each thread gets same seed, a different sequence number, no offset 
	// Original: Each thread gets same seed, a different sequence number, no offset
	//hiprand_init(1984, pixel_index, 0, &rand_state[pixel_index]);
	// BUGFIX, see Issue#2: Each thread gets different seed, same sequence for
	// performance improvement of about 2x!
	hiprand_init(1984+pixel_index, 0, 0, &rand_state[pixel_index]);
}

__global__ void render(vec3* fb, int max_x, int max_y,
					   int ns, camera** cam, hittable** world, 
					   hiprandState *rand_state)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	if ((i >= max_x) || (j >= max_y)) return;
	int pixel_index = (j * max_x + i);
	hiprandState local_rand_state = rand_state[pixel_index];
	vec3 col(0, 0, 0);
	for (int s = 0; s < ns; s++)
	{
		double u = double(i + hiprand_uniform(&local_rand_state)) / double(max_x);
		double v = double(j + hiprand_uniform(&local_rand_state)) / double(max_y);
		ray r = (*cam)->get_ray(u, v, &local_rand_state);
		col += ray_color(r, world, &local_rand_state);
	}
	// The state value will change? 
	rand_state[pixel_index] = local_rand_state; 
	col /= double(ns); 
	col[0] = sqrt(col[0]); 
	col[1] = sqrt(col[1]); 
	col[2] = sqrt(col[2]); 
	fb[pixel_index] = col; 
}

#define RND (hiprand_uniform(&local_rand_state)) 

__global__ void create_world(hittable** d_list, hittable** d_world, camera** d_camera, int nx, int ny, hiprandState *rand_state)
{
	if (threadIdx.x == 0 && blockIdx.x == 0)
	{
		hiprandState local_rand_state = *rand_state; 
		d_list[0] = new sphere(vec3(0, -1000, -1), 1000, new lambertian(color3(0.5, 0.5, 0.5)));

		int i = 1; 
		for (int a = -11; a < 11; a++)
		{
			for (int b = -11; b < 11; b++)
			{
				double choose_mat = RND; 
				vec3 center(a + RND, 0.2, b + RND);
				if (choose_mat < 0.8)
				{
					d_list[i++] = new sphere(center, 0.2, new lambertian(color3(RND * RND, RND * RND, RND * RND)));
				}
				else if (choose_mat < 0.95)
				{
					d_list[i++] = new sphere(center, 0.2, new metal(color3(0.5 * (1 + RND), 0.5 * (1 + RND), 0.5 * (1 + RND)), 0.5 * RND));
				}
				else
				{
					d_list[i++] = new sphere(center, 0.2, new dieletric(1.5));
				}
			}
		}

		// this is to replace the vector<hitable> and append 
		d_list[i++] = new sphere(vec3(0.0, 1.0, 0.0), 1.0, new dieletric(1.5));
		d_list[i++] = new sphere(vec3(-4.0, 1.0, 0.0), 1.0, new lambertian(color3(0.4, 0.2, 0.1)));
		d_list[i++] = new sphere(vec3(4.0, 1.0, 0.0), 1.0, new metal(color3(0.7, 0.6, 0.5), 0.0));
		*rand_state = local_rand_state; 
		*d_world = new hittable_list(d_list, 1+22*22+3);
		// 
		point3 lookfrom(13, 2, 3);
		point3 lookat(0, 0, 0); 
		vec3 vup(0, 1, 0); 
		double dist_to_focus = 10.0; // (lookfrom - lookat).length();
		double aperture = 0.1; 
		*d_camera = new camera(	lookfrom, lookat, vup, 20.0, double(nx)/double(ny), aperture, dist_to_focus);
	}
}

// cast to void** for cudaMemallocManaged 
__global__ void free_world(hittable** d_list, hittable** d_world, camera** d_camera)
{
	for (int i = 0; i < 1 + 22 * 22 + 3; i++)
	{
		// d_list is a hittable ptr. suppose it does not have mat_ptr. 
		// this requires to convert to sphere ptr ... this is pretty not manual 
		delete ((sphere*)d_list[i])->mat_ptr;
		delete d_list[i];
	}
	delete* (d_world);
	delete* (d_camera);
}


int main()
{
	// profiling 
	clock_t time0, time1, time2;
	time0 = clock();

	// Image 
	int ns = 500; 
	int nx = 1200;
	int ny = 600;
	int tx = 8;
	int ty = 8;

	std::cerr << "Rendering a " << nx << "x" << ny << " image ";
	std::cerr << "in " << tx << "x" << ty << " blocks" << std::endl;

	int num_pixels = nx * ny;

	// allocate FB 
	color3* fb;
	size_t fb_size = num_pixels * sizeof(color3);
	checkCudaErrors(hipMallocManaged((void**)&fb, fb_size));

	// allocate random state 
	hiprandState* d_rand_state; 
	checkCudaErrors(hipMalloc((void**)&d_rand_state, num_pixels * sizeof(hiprandState)));
	hiprandState* d_rand_state2;
	checkCudaErrors(hipMalloc((void**)&d_rand_state2, 1 * sizeof(hiprandState)));
	rand_init << <1, 1 >> > (d_rand_state2);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	// world 
	hittable** d_list; 
	int num_hitables = 1 + 22 * 22 + 3; 
	checkCudaErrors(hipMalloc((void**)&d_list, num_hitables * sizeof(hittable*)));
	hittable** d_world; 
	checkCudaErrors(hipMalloc((void**)&d_world, sizeof(hittable*)));
	camera** d_camera;
	checkCudaErrors(hipMalloc((void**)&d_camera, sizeof(camera*)));
	create_world << <1, 1 >> > (d_list, d_world, d_camera, nx, ny, d_rand_state2); 
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	// Render 
	dim3 blocks(nx / tx + 1, ny / ty + 1);
	dim3 threads(tx, ty);
	render_init << <blocks, threads >> > (nx, ny, d_rand_state);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
	render << <blocks, threads >> > (fb, nx, ny, ns, d_camera, d_world, d_rand_state); 
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	// profiling 
	time1 = clock();
	double timer_seconds = ((double)(time1 - time0)) ;
	std::cerr << "Cuda took " << timer_seconds << " msec." << std::endl;


	// Output FB as Image 
	std::cout << "P3\n" << nx << " " << ny << "\n255\n";
	for (int j = ny - 1; j >= 0; j--)
	{
		for (int i = 0; i < nx; i++)
		{
			size_t pixel_index = (j * nx + i);
			int ir = static_cast<int>(255.99 * fb[pixel_index].r());
			int ig = static_cast<int>(255.99 * fb[pixel_index].g());
			int ib = static_cast<int>(255.99 * fb[pixel_index].b());
			std::cout << ir << ' ' << ig << ' ' << ib << std::endl; 
		}
	}

	//clean up 
	checkCudaErrors(hipDeviceSynchronize()); 
	free_world << <1, 1 >> > (d_list, d_world, d_camera); 
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipFree(d_list));
	checkCudaErrors(hipFree(d_world));
	checkCudaErrors(hipFree(d_camera));
	checkCudaErrors(hipFree(d_rand_state));
	// why previous did not do this 
	checkCudaErrors(hipFree(fb));

	// useful for cuda-memcheck --leak-check full
	hipDeviceReset();

	// profiling 
	time2 = clock();
	timer_seconds = ((double)(time2 - time1)) / CLOCKS_PER_SEC;
	std::cerr << "Ouput took " << timer_seconds << " sec." << std::endl;

	std::cerr << "Done!" << std::endl;
}